#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 16;
const int blockSize = 16;

__global__ 
void mykernel(char *a, int *b) {
	a[threadIdx.x] += b[threadIdx.x];
}


int main(void) {
  char a[N] = "Hello \0\0\0\0\0\0";
  int b[N] = {15, 10, 6, 0, -11, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
   
  char *ad;
  int *bd;

  const int csize = N * sizeof(char);
  const int isize = N * sizeof(char);

  printf("%s", a);

  hipMalloc( (void**)&ad, csize );
  hipMalloc( (void**)&bd, isize);

  hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

  dim3 dimBlock( blockSize, 1 );
  dim3 dimGrid( 1, 1 );
  mykernel<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
  hipFree( ad );
  hipFree( bd );

  printf("%s\n", a);
  return 0;
} 
